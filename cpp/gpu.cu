#include <stdint.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <assert.h>

extern "C" void compute2d_acc(uint8_t **, int, int, uint8_t *, uint32_t *);

#define BLOCK_SIZE 256

__global__ void run(uint8_t **input, uint8_t *metric, uint32_t num_rows, uint32_t num_cols, uint32_t *block_idxs, uint32_t *output) {
  uint64_t index = blockIdx.x;
  uint32_t first_col = block_idxs[index * 2];
  uint32_t second_col = block_idxs[index * 2 + 1];
  uint8_t *col1 = input[first_col];
  uint8_t *col2 = input[second_col];
  uint32_t *our_output = output + index * 512;

  __shared__ uint32_t counts[512];
  for (int i = threadIdx.x; i < 512; i += blockDim.x) {
    counts[i] = 0;
  }
  __syncthreads();

  uint32_t *met_32 = (uint32_t *)metric;
  uint32_t *col1_32 = (uint32_t *)col1;
  uint32_t *col2_32 = (uint32_t *)col2;
  for (uint32_t i = threadIdx.x; i < num_rows / 4; i += blockDim.x) {
    uint32_t counts_idx = (col1_32[i] << 4) | col2_32[i];
    uint32_t cur_met = met_32[i];
    for (int j = 0; j < 4; j++) {
      uint8_t single_idx = counts_idx & 255;
      atomicAdd(&counts[2 * single_idx], cur_met & 255);
      atomicAdd(&counts[2 * single_idx + 1], 1);
      counts_idx >>= 8;
      cur_met >>= 8;
    }
  }
  for (uint32_t i = num_rows / 4 * 4 + threadIdx.x; i < num_rows; i += blockDim.x) {
    uint8_t counts_idx = (col1[i] << 4) | col2[i];
    atomicAdd(&counts[2 * counts_idx], metric[i]);
    atomicAdd(&counts[2 * counts_idx + 1], 1);
  }
  __syncthreads();

  for (int i = threadIdx.x; i < 512; i += blockDim.x) {
    our_output[i] = counts[i];
  }
}

void compute2d_acc(uint8_t **cols, int num_rows, int num_cols, uint8_t *metric, uint32_t *stats) {
  assert(hipSetDevice(0) == hipSuccess);

  struct timeval start, end, diff, total;
  gettimeofday(&start, 0);
  uint8_t **input_dev;
  assert(hipMalloc((void **) &input_dev, sizeof(uint8_t *) * num_cols) == hipSuccess);
  uint8_t **col_dev_ptrs = (uint8_t **)malloc(sizeof(uint8_t *) * num_cols);
  for (int i = 0; i < num_cols; i++) {
    assert(hipMalloc((void **) &col_dev_ptrs[i], num_rows) == hipSuccess);
    hipMemcpy(col_dev_ptrs[i], cols[i], num_rows, hipMemcpyHostToDevice);
  }
  hipMemcpy(input_dev, col_dev_ptrs, sizeof(uint8_t *) * num_cols, hipMemcpyHostToDevice);
  uint8_t *metric_dev;
  assert(hipMalloc((void **) &metric_dev, num_rows) == hipSuccess);
  hipMemcpy(metric_dev, metric, num_rows, hipMemcpyHostToDevice);
  gettimeofday(&end, 0);
  timersub(&end, &start, &total);

  int num_pairs = num_cols * (num_cols - 1) / 2;
  int block_idxs_size = sizeof(uint32_t) * 2 * num_pairs;
  uint32_t *block_idxs = (uint32_t *)malloc(block_idxs_size);
  int pair_cnt = 0;
  for (int i = 0; i < num_cols; i++) {
    for (int j = i + 1; j < num_cols; j++) {
      block_idxs[2 * pair_cnt] = i;
      block_idxs[2 * pair_cnt + 1] = j;
      pair_cnt++;
    }
  }
  uint32_t *block_idxs_dev;
  assert(hipMalloc((void **) &block_idxs_dev, block_idxs_size) == hipSuccess);
  hipMemcpy(block_idxs_dev, block_idxs, block_idxs_size, hipMemcpyHostToDevice);

  uint32_t *output_dev;
  int output_size = sizeof(uint32_t) * 512 * num_pairs;
  assert(hipMalloc((void **) &output_dev, output_size) == hipSuccess);

  gettimeofday(&start, 0);
  run<<<num_pairs, BLOCK_SIZE>>>(input_dev, metric_dev, num_rows, num_cols, block_idxs_dev, output_dev);
  assert(hipDeviceSynchronize() == hipSuccess);
  gettimeofday(&end, 0);
  timersub(&end, &start, &diff);
  printf("GPU accelerator time: %ld.%06ld\n", (long)diff.tv_sec, (long)diff.tv_usec);

  gettimeofday(&start, 0);
  hipMemcpy(stats, output_dev, output_size, hipMemcpyDeviceToHost);
  gettimeofday(&end, 0);
  timersub(&end, &start, &diff);
  timeradd(&diff, &total, &total);
  printf("GPU data transfer time: %ld.%06ld\n", (long)total.tv_sec, (long)total.tv_usec);

  hipFree(output_dev);
  hipFree(block_idxs_dev);
  hipFree(input_dev);
  for (int i = 0; i < num_cols; i++) {
    hipFree(col_dev_ptrs[i]);
  }
  hipFree(metric_dev);
  free(col_dev_ptrs);
  free(block_idxs);
}
